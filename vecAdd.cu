#include "hip/hip_runtime.h"
#include <stdio.h>
#include <ctime>
#include <malloc.h>
#include "vecAdd.h"
#include "cudaHeaders.h"

__global__ void vecAdd(float *a, float *b, float *c, int size) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    printf("Hello from block %d, thread %d\n", blockIdx.x, threadIdx.x);

    if (i < size)
        *(c + i) = *(a + i) + *(b + i);
}


int doVecAdd() {
    hipEvent_t begin, begin_kernel, stop_kernel, stop;
    hipEventCreate(&begin);
    hipEventCreate(&begin_kernel);
    hipEventCreate(&stop_kernel);
    hipEventCreate(&stop);

    float *a, *b, *c;
    int size = 10;


    a = (float *)malloc(sizeof(float) * size);
    b = (float *)malloc(sizeof(float) * size);
    c = (float *)malloc(sizeof(float) * size);

    int j = 0;
    for (int i = 0; i<size; i++) {
        *(a + i) = j++;
        *(b + i) = j++;
    }

    float *d_a, *d_b, *d_c;

    hipMalloc(&d_a, size*sizeof(float));
    hipMalloc(&d_b, size*sizeof(float));
    hipMalloc(&d_c, size*sizeof(float));

    hipEventRecord(begin);

    hipMemcpy(d_a, a, size*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size*sizeof(float), hipMemcpyHostToDevice);

    hipEventRecord(begin_kernel);

    vecAdd <<<1, 1024 >>>(d_a, d_b, d_c, size);

    hipEventRecord(stop_kernel);
    gpuErrchk(hipPeekAtLastError());

    hipMemcpy(c, d_c, size*sizeof(float), hipMemcpyDeviceToHost);

    hipEventRecord(stop);
    hipEventSynchronize(stop_kernel);
    hipEventSynchronize(stop);
    float kernelTime, totalTime; // Initialize elapsedTime;
    hipEventElapsedTime(&kernelTime, begin_kernel, stop_kernel);
    hipEventElapsedTime(&totalTime, begin, stop);
    printf("Time for KERNEL execution is: %fms\n", kernelTime);
    printf("Total time is: %fms\n", totalTime);

    printf("A vector:\n");
    for(int i=0; i<size; i++)
    printf("%.3f ", *(a+i));
    printf("\n");

    printf("B vector:\n");
    for(int i=0; i<size; i++)
    printf("%.3f ",*(b+i));
    printf("\n");

    printf("C vector:\n");
    for(int i=0; i<size; i++)
    printf("%.3f ", *(c+i));
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}
